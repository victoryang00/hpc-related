#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <moderngpu.cuh>

template<int NT, int VT, typename Key>
__global__ void BlockSortKernel(Key *d_in, Key *d_out)
{
    int tid = threadIdx.x;
    Key key[VT];

    __shared__ Key shared[NT*VT+1];

    mgpu::DeviceGlobalToThread<NT, VT>(NT*VT, d_in, tid, key);
    mgpu::CTAMergesortKeys<NT, VT, false>(key, shared, VT*NT, tid, mgpu::less<int>());
    mgpu::DeviceSharedToGlobal<NT, VT>(NT*VT, shared, tid, d_out);
}

int main(int argc, char *argv[])
{
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(argc, argv, true);

    const int vt = 2;
    MGPU_MEM(int32_t) d_in  = context->GenRandom<int32_t>(192 * vt, 1, 1000);
    MGPU_MEM(int32_t) d_out = context->Malloc<int32_t>(192 * vt);

    BlockSortKernel<192, vt, int32_t><<<1, 192>>>(*d_in, *d_out);

    // context->Start();
    // double elapsed = context->Split();

    // printf("Time elapsed: %.2f\n", elapsed);

    // puts("Input array: ");
    // mgpu::PrintArray(*d_in, "%6d", 12);
    // puts("Output array: ");
    // mgpu::PrintArray(*d_out, "%6d", 12);

    return 0;
}
