#include "hip/hip_runtime.h"
#define NO_CPP11

#include <iostream>
#include <moderngpu.cuh>
#include <queue>

#include "puzzle/GPU-solver.cuh"
#include "puzzle/GPU-kernel.cuh"

using namespace mgpu;

int div_up(int x, int y) { return (x-1) / y + 1; }

struct DeviceData {
    // store the structure of the grid graph
    MGPU_MEM(uint8_t) graph;

    // store open list + close list
    MGPU_MEM(node_t) nodes;
    MGPU_MEM(int) nodeSize;

    // hash table for `nodes'
    MGPU_MEM(uint32_t) hash;

    // store open list
    MGPU_MEM(heap_t) openList;
    // store the size for each heap
    MGPU_MEM(int) heapSize;
    MGPU_MEM(int) heapBeginIndex;

    MGPU_MEM(heap_t) heapInsertList;
    MGPU_MEM(int) heapInsertSize;

    // current shortest distance (a float)
    MGPU_MEM(uint32_t) optimalDistance;
    // store the result return by the GPU
    MGPU_MEM(heap_t) optimalNodes;
    // store the size for optimalNodes
    MGPU_MEM(int) optimalNodesSize;

    MGPU_MEM(uint32_t) lastAddr;
    MGPU_MEM(uint32_t) answerList;
    MGPU_MEM(int) answerSize;

    ContextPtr context;
};


GPUPathwaySolver::GPUPathwaySolver(Pathway *pathway)
    : p(pathway)
{
    d = new DeviceData();
}

GPUPathwaySolver::~GPUPathwaySolver()
{
    // vector<node_t> nodes;
    // vector<uint32_t> hash;
    // d->nodes->ToHost(nodes, d->nodeSize->Value());
    // d->hash->ToHost(hash, p->size());
    // for (;;) {
    //     cout << "(x, y): ";
    //     int x, y;
    //     int px, py;
    //     cin >> x >> y;
    //     int nodeID = p->toID(x, y);
    //     int hashValue = hash[nodeID];
    //     int prevID = nodes[nodes[hashValue].prev].nodeID;
    //     p->toXY(prevID, &px, &py);
    //     std::cout << "fValue: " << nodes[hashValue].fValue << endl
    //               << "gValue: " << nodes[hashValue].gValue << endl
    //               << "prev: " << px << ", " << py << endl << endl;;
    // }
    delete d;
}

void GPUPathwaySolver::initialize()
{
    hipDeviceSynchronize();
    hipDeviceReset();

        if (N == 3) {
            dbCount = 1;
            tracked.resize(dbCount);
            database.resize(dbCount);
            for (int i = 1; i <= 8; ++i)
                tracked[0].push_back(i);
        } else if (N == 4) {
            dbCount = 2;
            tracked.resize(dbCount);
            database.resize(dbCount);

            tracked[0].push_back(1);
            tracked[0].push_back(2);
            tracked[0].push_back(3);
            tracked[0].push_back(4);
            tracked[0].push_back(5);
            tracked[0].push_back(6);
            tracked[0].push_back(9);
            tracked[0].push_back(13);

            tracked[1].push_back(7);
            tracked[1].push_back(8);
            tracked[1].push_back(10);
            tracked[1].push_back(11);
            tracked[1].push_back(12);
            tracked[1].push_back(14);
            tracked[1].push_back(15);
        } else if (N == 5) {
            dbCount = 4;
            tracked.resize(dbCount);
            database.resize(dbCount);

            tracked[0].push_back(3);
            tracked[0].push_back(4);
            tracked[0].push_back(5);
            tracked[0].push_back(10);
            tracked[0].push_back(15);
            tracked[0].push_back(20);

            tracked[1].push_back(2);
            tracked[1].push_back(1);
            tracked[1].push_back(6);
            tracked[1].push_back(11);
            tracked[1].push_back(16);
            tracked[1].push_back(21);

            tracked[2].push_back(7);
            tracked[2].push_back(8);
            tracked[2].push_back(9);
            tracked[2].push_back(12);
            tracked[2].push_back(17);
            tracked[2].push_back(22);

            tracked[3].push_back(13);
            tracked[3].push_back(14);
            tracked[3].push_back(18);
            tracked[3].push_back(19);
            tracked[3].push_back(23);
            tracked[3].push_back(24);
        } else
            assert(false);

        index.resize(dbCount);
        multiple.resize(dbCount);
        mapTracked.resize(N*N, make_pair(-1, -1));
        for (int i = 0; i < dbCount; ++i) {
            PatternDatabase pd(N, tracked[i]);
            database[i].resize(pd.size());
            pd.fetchDatabase(database[i].data());

            index[i].resize(tracked[i].size());
            multiple[i].reserve(tracked[i].size() + 1);

            for (int j = 0; j < (int)tracked[i].size(); ++j) {
                mapTracked[tracked[i][j]] = make_pair(i, j);
                multiple[i].push_back(N*N - j);
            }
            multiple[i].push_back(1);

            for (int j = tracked[i].size()-2; j >= 0; --j)
                multiple[i][j] *= multiple[i][j+1];

            for (int j = 0; j <= tracked[i].size(); ++j)
                cout << multiple[i][j] << " ";
            cout << endl;
        }

        node_t<N> *node = new node_t<N>;
        vector<uint8_t> state;
        p->initialState(state);

        node->ps = PuzzleStorage<N>(
            *reinterpret_cast<uint8_t(*)[N][N]>(state.data()));
        node->prev = 0;
        node->fValue = computeHValue(
            *reinterpret_cast<uint8_t(*)[N][N]>(state.data()));
        node->gValue = 0;

        heap_t<N> heapItem;
        heapItem.fValue = node->fValue;
        heapItem.node = node;

        openList.push(heapItem);
        closeList[node->ps] = node;

        int count = 0;
        uint8_t _targetState[N][N];
        for (int i = 0; i < N; ++i)
            for (int j = 0; j < N; ++j)
                _targetState[i][j] = ++count;
        _targetState[N-1][N-1] = 0;
        targetState = PuzzleStorage<N>(_targetState);


    d->context = CreateCudaDevice(vm_options["ordinal"].as<int>());

    initializeCUDAConstantMemory(
        p->height(), p->width(), p->ex(), p->ey(),
        (uint32_t)p->toID(p->ex(), p->ey()));

    d->graph = d->context->Malloc<uint8_t>(p->graph(), p->size());

    d->nodes = d->context->Malloc<node_t>(NODE_LIST_SIZE);
    d->nodeSize = d->context->Fill<int>(1, 1);

    d->hash = d->context->Fill<uint32_t>(p->size(), UINT32_MAX);

    d->openList = d->context->Malloc<heap_t>(OPEN_LIST_SIZE);
    d->heapSize = d->context->Fill<int>(NUM_TOTAL, 0);
    d->heapBeginIndex = d->context->Fill<int>(1, 0);

    d->sortList = d->context->Malloc<sort_t>(NUM_VALUE * 8);
    d->prevList = d->context->Malloc<uint32_t>(NUM_VALUE * 8);
    d->sortList2 = d->context->Malloc<sort_t>(NUM_VALUE * 8);
    d->prevList2 = d->context->Malloc<uint32_t>(NUM_VALUE * 8);
    d->sortListSize = d->context->Fill<int>(1, 0);
    d->sortListSize2 = d->context->Fill<int>(1, 0);

    d->heapInsertList = d->context->Malloc<heap_t>(NUM_VALUE * 8);
    d->heapInsertSize = d->context->Fill<int>(1, 0);

    d->optimalDistance = d->context->Fill<uint32_t>(1, UINT32_MAX);
    d->optimalNodes = d->context->Malloc<heap_t>(NUM_TOTAL);
    d->optimalNodesSize = d->context->Fill<int>(1, 0);

    d->lastAddr = d->context->Malloc<uint32_t>(1);
    d->answerList = d->context->Malloc<uint32_t>(ANSWER_LIST_SIZE);
    d->answerSize = d->context->Fill<int>(1, 0);

    kInitialize<<<1, 1>>>(
        *d->nodes,
        *d->hash,
        *d->openList,
        *d->heapSize,
        p->sx(),
        p->sy()
    );
    dout << "\t\tGPU Initialization finishes" << endl;
}

bool GPUPathwaySolver::solve()
{
    std::priority_queue< heap_t, vector<heap_t>, std::greater<heap_t> > pq;

    for (int round = 0; ;++round) {
        if (DEBUG_CONDITION) {
            vector<int> heapSize;
            d->heapSize->ToHost(heapSize, NUM_TOTAL);
            printf("\t\t\t Heapsize: %d of %d\n", heapSize[0], HEAP_CAPACITY);
        }

        // printf("\t\tRound %d\n", round); fflush(stdout);
        dprintf("\t\tRound %d: kExtractExpand\n", round);
        kExtractExpand<
            NUM_BLOCK, NUM_THREAD, VALUE_PER_THREAD, HEAP_CAPACITY> <<<
            NUM_BLOCK, NUM_THREAD>>>(
                *d->nodes,

                *d->graph,

                *d->openList,
                *d->heapSize,

                *d->optimalDistance,
                *d->optimalNodes,
                *d->optimalNodesSize,

                *d->sortList,
                *d->prevList,
                *d->sortListSize,

                // reset them BTW
                *d->heapBeginIndex,
                *d->heapInsertSize
            );
#ifdef KERNEL_LOG
        hipDeviceSynchronize();
#endif

        dprintf("\t\tRound %d: Fetch optimalNodesSize: ", round);
        int optimalNodesSize = d->optimalNodesSize->Value();
        dprintf("%d\n", optimalNodesSize);

        if (optimalNodesSize) {
            printf("\t\tRound %d: Found one solution\n", round);
            vector<heap_t> optimalNodes;
            d->optimalNodes->ToHost(optimalNodes, optimalNodesSize);

            uint32_t optimalDistance = d->optimalDistance->Value();
            dprintf("\t\tRound %d: Fetch optimalDistance: %.2f\n", round, reverseFlipFloat(optimalDistance));

            for (size_t i = 0; i != optimalNodes.size(); ++i) {
                dprintf("\t\t\t optimalNodes[%d]: %.3f\n", (int)i, optimalNodes[i].fValue);
                pq.push(optimalNodes[i]);
            }

            dprintf("\t\t\t pq.top(): %.3f\n", pq.top().fValue);
            if (flipFloat(pq.top().fValue) <= optimalDistance) {
                printf("\t\t\t Number of nodes expanded: %d\n", d->nodeSize->Value());
                m_optimalNodeAddr = pq.top().addr;
                m_optimalDistance = pq.top().fValue;
                dprintf("\t\t\t Optimal nodes address: %d\n", m_optimalNodeAddr);
                return true;
            }
        }

        dprintf("\t\tRound %d: Fetch sortListSize: ", round);
        int sortListSize = d->sortListSize->Value();
        dprintf("%d\n", sortListSize);
        // if (round % 2000 == 0) {
        //     printf("\t\tRound %d: Fetch sortListSize: %d\n", round, sortListSize);
        // }
        if (sortListSize == 0)
            return false;

        dprintf("\t\tRound %d: MergesortPairs\n", round);
        MergesortPairs(
            d->sortList->get(),
            d->prevList->get(),
            sortListSize,
            *d->context
        );

        dprintf("\t\tRound %d: kAssign\n", round);
        kAssign<NUM_THREAD><<<
            div_up(sortListSize, NUM_THREAD), NUM_THREAD>>> (
                *d->sortList,
                *d->prevList,
                sortListSize,

                *d->sortList2,
                *d->prevList2,
                *d->sortListSize2
            );
#ifdef KERNEL_LOG
        hipDeviceSynchronize();
#endif

        dprintf("\t\tRound %d: Fetch sortListSize2: ", round);
        int sortListSize2 = d->sortListSize2->Value();
        dprintf("%d\n", sortListSize2);
        // if (round % 2000 == 0) {
        //     printf("\t\tRound %d: Fetch sortListSize2: %d\n", round, sortListSize2);
        // }

        dprintf("\t\tRound %d: kDeduplicate\n", round);
        // printf("\t\tRound %d: nodeSize: %d\n", round, d->nodeSize->Value());
        kDeduplicate<NUM_THREAD> <<<
            div_up(sortListSize2, NUM_THREAD), NUM_THREAD>>> (
                *d->nodes,
                *d->nodeSize,

                *d->hash,

                *d->sortList2,
                *d->prevList2,
                sortListSize2,

                *d->heapInsertList,
                *d->heapInsertSize
            );
        // printf("\t\tRound %d: nodeSize: %d\n", round, d->nodeSize->Value());
#ifdef KERNEL_LOG
        hipDeviceSynchronize();
#endif

        dprintf("\t\tRound %d: kHeapInsert\n", round);
        kHeapInsert<
            NUM_BLOCK, NUM_THREAD, HEAP_CAPACITY> <<<
            NUM_BLOCK, NUM_THREAD>>> (
                *d->openList,
                *d->heapSize,
                *d->heapBeginIndex,

                *d->heapInsertList,
                *d->heapInsertSize,

                // reset them BTW
                *d->sortListSize,
                *d->sortListSize2,
                *d->optimalDistance,
                *d->optimalNodesSize
            );
#ifdef KERNEL_LOG
        hipDeviceSynchronize();
#endif
        dprintf("\t\tRound %d: Finished\n\n", round);
    }
}

void GPUPathwaySolver::getSolution(float *optimal, vector<int> *pathList)
{
    d->lastAddr->FromHost(&m_optimalNodeAddr, 1);
    kFetchAnswer<<<1, 1>>>(
        *d->nodes,

        *d->lastAddr,

        *d->answerList,
        *d->answerSize
    );

    int answerSize = d->answerSize->Value();

    vector<uint32_t> answerList;
    d->answerList->ToHost(answerList, answerSize);

    *optimal = m_optimalDistance;
    pathList->clear();
    pathList->reserve(answerSize);
    for (int i = answerSize-1; i >= 0; --i) {
        pathList->push_back((int)answerList[i]);
    }

}

bool GPUPathwaySolver::isPrime(uint32_t number)
{
    uint32_t upper = sqrt(number) + 1;
    assert(upper < number);

    for (uint32_t i = 2; i != upper; ++i)
        if (number % i == 0)
            return false;
    return true;
}

vector<uint32_t> GPUPathwaySolver::genRandomPrime(uint32_t maximum, int count)
{
    vector<uint32_t> result;
    int prepare = 3 * count;

    uint32_t now = maximum;
    while (prepare) {
        if (isPrime(now))
            result.push_back(now);
        now--;
    }

    std::random_shuffle(result.begin(), result.end());
    result.erase(result.begin() + count, result.end());

    for (int i = 0; i < count; ++i)
        dout << result[i] << " ";
    dout << endl;

    return result;
}
